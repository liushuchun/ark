#include "hip/hip_runtime.h"
/*!
 * Copyright (c) 2015 by Contributors
 * \file activation.cu
 * \brief
 * \author Bing Xu
*/
#include "./activation-inl.h"
#include "./mshadow_op.h"
#if MXNET_USE_CUDNN == 1
#include "./cudnn_activation-inl.h"
#endif

namespace mxnet {
namespace op {
template<>
Operator *CreateOp<gpu>(ActivationParam param) {
#if MXNET_USE_CUDNN == 1
  return new CuDNNActivationOp(param);
#else
  switch(param.act_type) {
    case activation::kReLU:
      return new ActivationOp<gpu, mshadow_op::relu, mshadow_op::relu_grad>();
    case activation::kSigmoid:
      return new ActivationOp<gpu, mshadow_op::sigmoid, mshadow_op::sigmoid_grad>();
    case activation::kTanh:
      return new ActivationOp<gpu, mshadow_op::tanh, mshadow_op::tanh_grad>();
    default:
      LOG(FATAL) << "unknown activation";
      return NULL;
  }
#endif  // MXNET_USE_CUDNN
}
}  // op
}  // namespace mxnet

